#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>

#define THREADSPERBLOCK 8

__global__ void multiplicaMatriz(double *matrizACuda, double *matrizBCuda, double *matrizCCuda, int tam) 
{
    /*Utilizamos 2 sub matrizes, que irão armazenar os valores das matrizes A e B necessários
    para nosso cálculo, essas variáveis são compartilhadas entre as threads*/
    __shared__ double subMatrizA[THREADSPERBLOCK][THREADSPERBLOCK];
    __shared__ double subMatrizB[THREADSPERBLOCK][THREADSPERBLOCK];

    //Calculamos qual linha é de nossa responsabilidade
    int linha = blockIdx.x * THREADSPERBLOCK + threadIdx.x;
    //Calculamos qual coluna é de nossa responsabilidade
    int coluna = blockIdx.y * THREADSPERBLOCK + threadIdx.y;
    
    //Variavel que armazenará o valor calculado
    double calculo = 0;


    //Faz o calculo o número de vezes necessário (basicamente iterando entre as grades)
    for (int pulo = 0; pulo < gridDim.x; ++pulo) 
    {
        //Calcula a posição do valor que iremos pegar da matriz A
        int idx = linha * tam + pulo * THREADSPERBLOCK + threadIdx.y;
        //Se a posição ultrapassar o limite, apenas colocamos 0 em nossa sub matriz
        if(idx >= tam*tam)
            subMatrizA[threadIdx.x][threadIdx.y] = 0;
        //Caso contrário colocamos o valor em nossa sub matriz
        else
            subMatrizA[threadIdx.x][threadIdx.y] = matrizACuda[idx];
        //Calcula a posição do valor que iremos pegar da matriz B
        idx = (pulo * THREADSPERBLOCK + threadIdx.x) * tam + coluna;
        //Se a posição ultrapassar o limite, apenas colocamos 0 em nossa sub matriz
        if(idx >= tam*tam)
            subMatrizB[threadIdx.x][threadIdx.y] = 0;
        //Caso contrário colocamos o valor em nossa sub matriz
        else
            subMatrizB[threadIdx.x][threadIdx.y] = matrizBCuda[idx];

        //É necessário haver uma sincronização das threads para somarmos a resposta, por conta de nossas variáveis compartilhadas
        __syncthreads();
        //É feito o calculo do valor
        for (int k = 0; k < THREADSPERBLOCK; ++k) 
            calculo += subMatrizA[threadIdx.x][k] * subMatrizB[k][threadIdx.y];

        //Aguarda as threads sincronizarem novamente antes de começar uma nova iteração
        __syncthreads();
    }
    //Se estiver tudo correto com nossos indices de linha e coluna atualizamos o valor da matriz resultado C
    if(linha < tam && coluna < tam)
        matrizCCuda[linha * tam + coluna] = calculo;
}


int main(int argc,char **argv){
    //Declara as matrizes que irão para a GPU
    double *matrizACuda,*matrizBCuda,*matrizCCuda;
    //Declara as matrizes que ficarão na CPU
    double *matrizA,*matrizB,*matrizC; 
    //Declara as variáveis de tamanho e índice
    int tam,i,j;

    //Lê a dimensão da matriz
    fscanf(stdin,"%d\n",&tam); 

    //Aloca as matrizes do host
    hipHostMalloc((void**)&matrizA,tam*tam*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&matrizB,tam*tam*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&matrizC,tam*tam*sizeof(double), hipHostMallocDefault);
    //Aloca as matrizes do device
    hipMalloc((void **) &matrizACuda, sizeof(double)*tam*tam);
    hipMalloc((void **) &matrizBCuda, sizeof(double)*tam*tam);
    hipMalloc((void **) &matrizCCuda, sizeof(double)*tam*tam);

    //Lê as matrizes A e B
    for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            fscanf(stdin, "%lf ", &matrizA[i * tam + j]);
    for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            fscanf(stdin,"%lf ",&matrizB[i*tam+j]);
    
    //Envia do host para o Device
    hipMemcpy(matrizACuda, matrizA, tam*tam*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(matrizBCuda, matrizB, tam*tam*sizeof(double),hipMemcpyHostToDevice);
    
    //Calcula a quantidade de blocos por grade (haverão tam threads por grade)
    int blocksPerGrid = (tam+THREADSPERBLOCK-1)/THREADSPERBLOCK;
    //Define nossas threads e nossos blocos
    dim3 dimGrid(blocksPerGrid,blocksPerGrid);
    dim3 dimBlock(THREADSPERBLOCK,THREADSPERBLOCK);

    //Chama a função para fazer a multiplicação
    multiplicaMatriz<<<dimGrid, dimBlock>>>(matrizACuda, matrizBCuda, matrizCCuda, tam);

    //Envia a resposta do Device para o Host
    hipMemcpy(matrizC, matrizCCuda, sizeof(double)*tam*tam, hipMemcpyDeviceToHost);

    //Imprime o resultado    
    for(i=0;i<tam;i++){
        for(j=0;j<tam;j++)
            printf("%.1lf ",matrizC[i*tam+j]);
        printf("\n");
    }
    
    //Desaloca as matrizes do device
    hipFree(matrizACuda);
    hipFree(matrizBCuda);
    hipFree(matrizCCuda);
    //Desaloca as matrizes do host
    hipHostFree(matrizA);
    hipHostFree(matrizB);
    hipHostFree(matrizC);

    return 0;
}
