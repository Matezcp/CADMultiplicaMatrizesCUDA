#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>

#define THREADSPERBLOCK 8

__global__ void multiplicaMatriz(double *matrizACuda, double *matrizBCuda, double *matrizCCuda, int tam) 
{
    /*Utilizamos 2 sub matrizes, que irão armazenar os valores das matrizes A e B necessários
    para nosso cálculo, essas variáveis são compartilhadas entre as threads*/
    __shared__ double subMatrizA[THREADSPERBLOCK][THREADSPERBLOCK];
    __shared__ double subMatrizB[THREADSPERBLOCK][THREADSPERBLOCK];

    //Calculamos qual linha é de nossa responsabilidade
    int linha = blockIdx.y * THREADSPERBLOCK + threadIdx.y;
    //Calculamos qual coluna é de nossa responsabilidade
    int coluna = blockIdx.x * THREADSPERBLOCK + threadIdx.x;
    
    //Variavel que armazenará o valor calculado
    double calculo = 0;

    __syncthreads();
    printf("----------------------------\nINFOS: gridim.x: %d blockidx.x: %d blockIdx.y: %d threadIdx.x: %d threadIdx.y: %d Linha: %d coluna: %d\n",gridDim.x,blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,linha,coluna);

    //Faz os calculos blocksPerGrid vezes 
    for (int pulo = 0; pulo < gridDim.x; ++pulo) 
    {
        //Calcula a posição do valor que iremos pegar da matriz A
        int idx = linha * tam + pulo * THREADSPERBLOCK + threadIdx.x;
        //Se a posição ultrapassar o limite, apenas colocamos 0 em nossa sub matriz
        if(idx >= tam*tam)
            subMatrizA[threadIdx.y][threadIdx.x] = 0;
        //Caso contrário colocamos o valor em nossa sub matriz
        else
            subMatrizA[threadIdx.y][threadIdx.x] = matrizACuda[idx];
        //Calcula a posição do valor que iremos pegar da matriz B
        idx = (pulo * THREADSPERBLOCK + threadIdx.y) * tam + coluna;
        //Se a posição ultrapassar o limite, apenas colocamos 0 em nossa sub matriz
        if(idx >= tam*tam)
            subMatrizB[threadIdx.y][threadIdx.x] = 0;
        //Caso contrário colocamos o valor em nossa sub matriz
        else
            subMatrizB[threadIdx.y][threadIdx.x] = matrizBCuda[idx];

        //É necessário haver uma sincronização das threads para somarmos a resposta, por conta de nossas variáveis compartilhadas
        __syncthreads();
        //É feito o calculo do valor
        for (int k = 0; k < THREADSPERBLOCK; ++k) 
            calculo += subMatrizA[threadIdx.y][k] * subMatrizB[k][threadIdx.x];

        //Aguarda as threads sincronizarem novamente antes de começar uma nova iteração
        __syncthreads();
    }
    //Se estiver tudo correto com nossos indices de linha e coluna atualizamos o valor da matriz resultado C
    if(linha < tam && coluna < tam)
        matrizCCuda[linha * tam + coluna] = calculo;
}


int main(int argc,char **argv){
    //Declara as matrizes que irão para a GPU
    double *matrizACuda,*matrizBCuda,*matrizCCuda;
    //Declara as matrizes que ficarão na CPU
    double *matrizA,*matrizB,*matrizC; 
    //Declara as variáveis de tamanho e índice
    int tam,i,j;

    //Lê a dimensão da matriz
    fscanf(stdin,"%d\n",&tam); 

    //Aloca as matrizes do host
    hipHostMalloc((void**)&matrizA,tam*tam*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&matrizB,tam*tam*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&matrizC,tam*tam*sizeof(double), hipHostMallocDefault);
    //Aloca as matrizes do device
    hipMalloc((void **) &matrizACuda, sizeof(double)*tam*tam);
    hipMalloc((void **) &matrizBCuda, sizeof(double)*tam*tam);
    hipMalloc((void **) &matrizCCuda, sizeof(double)*tam*tam);

    //Lê as matrizes A e B
    for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            fscanf(stdin, "%lf ", &matrizA[i * tam + j]);
    for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            fscanf(stdin,"%lf ",&matrizB[i*tam+j]);
    
    //Envia do host para o Device
    hipMemcpy(matrizACuda, matrizA, tam*tam*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(matrizBCuda, matrizB, tam*tam*sizeof(double),hipMemcpyHostToDevice);
    
    //Calcula a quantidade de blocos por grade (haverão tam threads por grade)
    int blocksPerGrid = (tam+THREADSPERBLOCK-1)/THREADSPERBLOCK;
    //Define nossas threads e nossos blocos
    dim3 dimGrid(blocksPerGrid,blocksPerGrid);
    dim3 dimBlock(THREADSPERBLOCK,THREADSPERBLOCK);

    //Chama a função para fazer a multiplicação
    multiplicaMatriz<<<dimGrid, dimBlock>>>(matrizACuda, matrizBCuda, matrizCCuda, tam);

    //Envia a resposta do Device para o Host
    hipMemcpy(matrizC, matrizCCuda, sizeof(double)*tam*tam, hipMemcpyDeviceToHost);

    //Imprime o resultado    
    for(i=0;i<tam;i++){
        for(j=0;j<tam;j++)
            printf("%.1lf ",matrizC[i*tam+j]);
        printf("\n");
    }
    
    //Desaloca as matrizes do device
    hipFree(matrizACuda);
    hipFree(matrizBCuda);
    hipFree(matrizCCuda);
    //Desaloca as matrizes do host
    hipHostFree(matrizA);
    hipHostFree(matrizB);
    hipHostFree(matrizC);

    return 0;
}
