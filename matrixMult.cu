#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void gpu_square_matrix_mult(double *matrizACuda, double *matrizBCuda, double *matrizCCuda, int n) 
{
    __shared__ double tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int linha = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int coluna = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = linha * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = matrizACuda[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + coluna;
        if(idx >= n*n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }  
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = matrizBCuda[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(linha < n && coluna < n)
    {
        matrizCCuda[linha * n + coluna] = tmp;
    }
}


int main(int argc,char **argv){
    //Declara as matrizes que irão para a GPU
    double *matrizACuda,*matrizBCuda,*matrizCCuda;
    //Declara as matrizes que ficarão na CPU
    double *matrizA,*matrizB,*matrizC; 
    //Declara as variáveis de tamanho e índice
    int tam,i,j,k; 

    //Lê a dimensão da matriz
    fscanf(stdin,"%d\n",&tam); 

    //Aloca as matrizes
    matrizA=(double*)malloc(tam*tam*sizeof(double));
    matrizB=(double*)malloc(tam*tam*sizeof(double));
    matrizC=(double*)malloc(tam*tam*sizeof(double));
    hipMalloc((void **) &matrizACuda, sizeof(double)*tam*tam);
    hipMalloc((void **) &matrizBCuda, sizeof(double)*tam*tam);
    hipMalloc((void **) &matrizCCuda, sizeof(double)*tam*tam);

    //Lê as matrizes A e B
    for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            fscanf(stdin, "%lf ", &matrizA[i * tam + j]);
    for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            fscanf(stdin,"%lf ",&matrizB[i*tam+j]);
    
    //Envia do host para o Device
    hipMemcpy(matrizACuda, matrizA, tam*tam*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(matrizBCuda, matrizB, tam*tam*sizeof(double),hipMemcpyHostToDevice);

    //Calcula C=A*B
    /*for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            for(k=0;k<tam;k++)
                matrizC[i*tam+j]+=matrizA[i*tam+k]*matrizB[k*tam+j];*/
    
    //Calcula a carga de trabalho
    unsigned int carga_trabalho = (tam + BLOCK_SIZE - 1) / BLOCK_SIZE;
    //Define nossas threads e nossos blocos
    dim3 dimGrid(carga_trabalho, carga_trabalho);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    //Chama a função para fazer a multiplicação
    gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(matrizACuda, matrizBCuda, matrizCCuda, tam);    

    //Envia a resposta do Device para o Host
    hipMemcpy(matrizC, matrizCCuda, sizeof(double)*tam*tam, hipMemcpyDeviceToHost);

    //Imprime o resultado    
    for(i=0;i<tam;i++){
        for(j=0;j<tam;j++)
            printf("%.1lf ",matrizC[i*tam+j]);
        printf("\n");
    }
    
    //Desaloca as matrizes
    hipFree(matrizACuda);
    hipFree(matrizBCuda);
    hipFree(matrizCCuda);
    hipHostFree(matrizA);
    hipHostFree(matrizB);
    hipHostFree(matrizC);

    return 0;
}
