#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void multiplicaMatriz(double *matrizACuda, double *matrizBCuda, double *matrizCCuda, int n) 
{
    __shared__ double subMatrizA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double subMatrizB[BLOCK_SIZE][BLOCK_SIZE];

    int linha = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int coluna = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int idx;
    double calculo = 0;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = linha * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
            subMatrizA[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            subMatrizA[threadIdx.y][threadIdx.x] = matrizACuda[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + coluna;
        if(idx >= n*n)
        {
            subMatrizB[threadIdx.y][threadIdx.x] = 0;
        }  
        else
        {
            subMatrizB[threadIdx.y][threadIdx.x] = matrizBCuda[idx];
        }
        __syncthreads();
        
        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            calculo += subMatrizA[threadIdx.y][k] * subMatrizB[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(linha < n && coluna < n)
    {
        matrizCCuda[linha * n + coluna] = calculo;
    }
}


int main(int argc,char **argv){
    //Declara as matrizes que irão para a GPU
    double *matrizACuda,*matrizBCuda,*matrizCCuda;
    //Declara as matrizes que ficarão na CPU
    double *matrizA,*matrizB,*matrizC; 
    //Declara as variáveis de tamanho e índice
    int tam,i,j,k; 

    //Lê a dimensão da matriz
    fscanf(stdin,"%d\n",&tam); 

    //Aloca as matrizes
    matrizA=(double*)malloc(tam*tam*sizeof(double));
    matrizB=(double*)malloc(tam*tam*sizeof(double));
    matrizC=(double*)malloc(tam*tam*sizeof(double));
    hipMalloc((void **) &matrizACuda, sizeof(double)*tam*tam);
    hipMalloc((void **) &matrizBCuda, sizeof(double)*tam*tam);
    hipMalloc((void **) &matrizCCuda, sizeof(double)*tam*tam);

    //Lê as matrizes A e B
    for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            fscanf(stdin, "%lf ", &matrizA[i * tam + j]);
    for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            fscanf(stdin,"%lf ",&matrizB[i*tam+j]);
    
    //Envia do host para o Device
    hipMemcpy(matrizACuda, matrizA, tam*tam*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(matrizBCuda, matrizB, tam*tam*sizeof(double),hipMemcpyHostToDevice);

    //Calcula C=A*B
    /*for(i=0;i<tam;i++)
        for(j=0;j<tam;j++)
            for(k=0;k<tam;k++)
                matrizC[i*tam+j]+=matrizA[i*tam+k]*matrizB[k*tam+j];*/
    
    //Calcula a carga de trabalho
    unsigned int carga_trabalho = (tam + BLOCK_SIZE - 1) / BLOCK_SIZE;
    //Define nossas threads e nossos blocos
    dim3 dimGrid(carga_trabalho, carga_trabalho);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    //Chama a função para fazer a multiplicação
    multiplicaMatriz<<<dimGrid, dimBlock>>>(matrizACuda, matrizBCuda, matrizCCuda, tam);    

    //Envia a resposta do Device para o Host
    hipMemcpy(matrizC, matrizCCuda, sizeof(double)*tam*tam, hipMemcpyDeviceToHost);

    //Imprime o resultado    
    for(i=0;i<tam;i++){
        for(j=0;j<tam;j++)
            printf("%.1lf ",matrizC[i*tam+j]);
        printf("\n");
    }
    
    //Desaloca as matrizes
    hipFree(matrizACuda);
    hipFree(matrizBCuda);
    hipFree(matrizCCuda);
    hipHostFree(matrizA);
    hipHostFree(matrizB);
    hipHostFree(matrizC);

    return 0;
}
